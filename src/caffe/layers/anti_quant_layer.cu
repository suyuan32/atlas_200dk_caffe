#include "hip/hip_runtime.h"
/*
 * Copyright (c) Huawei Technologies Co., Ltd. 2019-2019. All rights reserved.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the Apache License Version 2.0.You may not use this file except in compliance with the License.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * Apache License for more details at
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * @brief anti_quant layer cuda src file
 *
 * @file anti_quant_layer.cu
 *
 * @version 1.0
 */
#include <vector>
#include <cmath>
#include <fstream>
#include <iostream>
#include <map>
#include "caffe/layers/anti_quant_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/amct_util.hpp"

namespace caffe {
template <typename Dtype>
void AntiQuantLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>* >& bottom,
                                        const vector<Blob<Dtype>* >& top)
{
    const int count = bottom[0]->count();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* mutable_top_data = top[0]->mutable_gpu_data();
    caffe_copy(count, bottom_data, mutable_top_data);
    int recordNum = bottom[0]->channels() * bottom[0]->height() * bottom[0]->width();
    if (recordData_) {
        std::string layerName = this->layer_param_.name();
        ConvertLayerName(layerName, "/", REPLACE_STR);
        std::string fileName = "./amct_log/" + layerName + ".log";
        INIT_LOG();
        DEBUG_LOG(fileName.c_str(), "[AMCT] bottom data of dequant:\n");
        DATA_DEBUG_LOG(fileName.c_str(), bottom[0]->cpu_data(), recordNum);
        DEBUG_LOG(fileName.c_str(), "[AMCT] top data of dequant:\n");
        DATA_DEBUG_LOG(fileName.c_str(), top[0]->cpu_data(), recordNum);
        recordData_ = false;
    }
}

template <typename Dtype>
void AntiQuantLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>* >& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>* >& bottom)
{
    const int count = bottom[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

    caffe_copy(count, top_diff, bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(AntiQuantLayer);
}  // namespace caffe
